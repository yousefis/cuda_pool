#include "hip/hip_runtime.h"
/*
 * Developer: Sahar Yousefi
 * more info: https://github.com/yousefis/cuda_pool
 */

#include"../include/kernels.cuh"
#include<iostream>
__global__
void sobel_filter(uchar* img1, uchar* result, size_t height, size_t width)
{
    int row = threadIdx.x + blockDim.x * blockIdx.x;
    int col = threadIdx.y + blockDim.y * blockIdx.y;

    if (row>=width) return;
    if (col>=height) return;
    float Gx = float(img1[(row-1)*height+(col-1)]) - float(img1[(row-1)*height+(col+1)])+
         2 * float(img1[row*height+(col-1)]) - 2 * float(img1[row*height+(col+1)])+
         float(img1[(row+1)*height+(col-1)]) - float(img1[(row+1)*height+(col+1)]);
    float Gy = float(img1[(row-1)*height+(col-1)]) + 2 * float(img1[(row-1)*height+col]) + float(img1[(row-1)*height+(col+1)])+
               (-1) * float(img1[(row+1)*height+(col-1)]) + (-2) * float(img1[(row+1)*height+col]) + (-1) * float(img1[(row+1)*height+(col+1)]);
    result[row*height + col] = sqrt (Gx*Gx + Gy*Gy);
        
}